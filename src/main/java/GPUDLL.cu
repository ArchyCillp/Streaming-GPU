#include "hip/hip_runtime.h"
#include "cn_edu_sustech_dbgroup_GPU_GPUNativeLib.h"
#include <math.h>
#include <hipcub/hipcub.hpp>
#include <assert.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define THREAD_PER_BLOCK 1024

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void
arrayComplexify(jlong *A, jdouble *B, jdouble *result, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        B[i] = sqrt(sin(fabs(cos((jdouble)A[i]))));
        // B[i] = 123.0;
    }
    if (i == 0) {
        result[0] = 0;
    }

    __syncthreads();

    typedef hipcub::BlockReduce<double, THREAD_PER_BLOCK> BlockReduceT; 
    __shared__ typename BlockReduceT::TempStorage temp_storage; 
    jdouble res;
    if (i < numElements) {
        res = BlockReduceT(temp_storage).Sum(B[i]);
        if (threadIdx.x == 0) {
            atomicAdd(result, res);
        }    
    }
}

JNIEXPORT jdouble JNICALL Java_cn_edu_sustech_dbgroup_GPU_GPUNativeLib_calculateArrayComplexSum
  (JNIEnv * env_, jobject this_, jlongArray arr_){
    jsize len = (*env_).GetArrayLength(arr_);
    jdouble sum = 0;
    jlong *body = (*env_).GetLongArrayElements(arr_, 0);

    jlong *d_body = NULL;
    int size = len * sizeof(jlong);
    gpuErrchk(hipMalloc((void**)&d_body, size));
    gpuErrchk(hipMemcpy(d_body, body, size, hipMemcpyHostToDevice));

    jdouble *d_tmp = NULL;
    int tmp_size = len * sizeof(jdouble);
    gpuErrchk(hipMalloc((void**)&d_tmp, tmp_size));

    jdouble *res = (jdouble *)malloc(sizeof(jdouble));
    jdouble *d_res = NULL;
    gpuErrchk(hipMalloc((void**)&d_res, sizeof(jdouble)));

    int threadsPerBlock = THREAD_PER_BLOCK;
    int blocksPerGrid = (len + threadsPerBlock - 1) / threadsPerBlock;

    arrayComplexify<<<blocksPerGrid, threadsPerBlock>>>(d_body, d_tmp, d_res, len);
    
    hipError_t err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    gpuErrchk(hipMemcpy(res, d_res, sizeof(jdouble), hipMemcpyDeviceToHost));


//DEBUG
    // jdouble *tmp = (jdouble *)malloc(sizeof(jdouble));
    // hipMemcpy(tmp, d_tmp, tmp_size, hipMemcpyDeviceToHost);
    // for (int i = 0; i < len; i++) {
    //     printf("%lf\n", tmp[i]);
    // }
    // free(tmp);

//--

    sum = res[0];

    hipFree(d_body);
    hipFree(d_tmp);
    hipFree(d_res);

    (*env_).ReleaseLongArrayElements(arr_, body, 0);
    return sum;
  }
